#include "hip/hip_runtime.h"
// PLEASE MODIFY THIS FILE TO IMPLEMENT YOUR SOLUTION

// Brute Force APSP Implementation:

#include "apsp.h"
#include "cuda_utils.h"
#include  <iostream>
#define b 32
#define MAX_SHARE_SIZE 32 * 1024 // real max 48KB

namespace {

__global__ void kernel(int n, int k, int *graph) {
    auto i = blockIdx.y * blockDim.y + threadIdx.y;
    auto j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n && j < n) {
        graph[i * n + j] = min(graph[i * n + j], graph[i * n + k] + graph[k * n + j]);
    }
}

}
__device__ void printGraph(int* graph, int n) {
    for(int j = 0; j < 2; j++) {
        for(int i = 0; i < 2; i++) {
            printf("i, j(%d, %d): %d \n", i, j, graph[j * n + i]);
        }
        // printf("xxxx\n");
    }
}

__global__ void kernel_stage2(int p, int n, int* graph) { // 第p步的操作。

    int centra_left = p * b;
    int centra_right = min((p + 1) * b, n);

    __shared__ int central_block[b][b]; // 中心块
    __shared__ int block[b][b]; // 待更新块
    
    int thread_x = threadIdx.x; // 块内偏移量
    int thread_y = threadIdx.y;

    int cent_x = thread_x + centra_left; // thread_x:[0, 32)
    int cent_y = thread_y + centra_left; // thread_y: [0, 32)
    if(cent_x < n && cent_y < n) { // 缓存中心块
        central_block[thread_x][thread_y] = graph[cent_x * n + cent_y];
    }

    // 缓存待更新块
    int block_x = blockIdx.x; // 对标centra_left
    int block_y = blockIdx.y;
    int i = -1;
    int j = -1;
    if(block_x >= p) block_x += 1; // 跳过中心块
    if(block_y == 1) { // 竖着的
        i = cent_x; // x方向和中心块一致
        j = thread_y + block_x * b;
        if(i < n && j < n) { // 缓存中心块
            block[thread_x][thread_y] = graph[i * n + j];
        }
    } else { // 横着的
        i = thread_x + block_x * b;
        j = cent_y;
        if(i < n && j < n) {
            block[thread_x][thread_y] = graph[i * n + j];
        }
    }
    // printf("p: %d; (i, j): (%d, %d)\n", p, i, j);

    __syncthreads(); // 同步

    // block[thread_x * n + thread_y] = std::min(block[thread_x * n + thread_y], block[thread_x * n + thread_y])
    int range = centra_right - centra_left;
    for(int k = 0; k < range; k++) {
        int ori_blcok = block[thread_x][thread_y];
        int tmp = central_block[thread_x][k] + block[k][thread_y];
        if(block_y == 1) {
            block[thread_x][thread_y] = min(block[thread_x][thread_y], central_block[thread_x][k] + block[k][thread_y]);
            if(ori_blcok != block[thread_x][thread_y]) {
                printf("vertical, threadx:%d, thready: %d\n", thread_x, thread_y);
                printf("k: %d, p: %d,(x, y): %d, %d, ori block not equal: %d, %d, x_k: %d, k_y: %d\n",k, p,
                 i, j, ori_blcok, block[thread_x][thread_y], central_block[thread_x][k], block[k][thread_y]);
            }
        }
        else {
            block[thread_x][thread_y] = min(block[thread_x][thread_y], block[thread_x][k] + central_block[k][thread_y]);
            if(ori_blcok != block[thread_x][thread_y]) {
                printf("horizonal, threadx:%d, thready: %d\n", thread_x, thread_y);
                printf("k: %d, p: %d,(x, y): %d, %d, ori block not equal: %d, %d, x_k: %d, k_y: %d\n",k, p,
                 i, j, ori_blcok, block[thread_x][thread_y], block[thread_x][k], central_block[k][thread_y]);
            }
        }
        __syncthreads();
    }
    // 写回
    if(block_y == 1) { // 竖着的
        int i = cent_x; // x方向和中心块一致
        int j = thread_y + block_x * b;
        if(i < n && j < n) { // 缓存中心块
            int ori_graph = graph[i * n + j];
            graph[i * n + j] = block[thread_x][thread_y];
            if(ori_graph != graph[i * n + j]) {
                printf("ori block not equalxxxx: %d, %d\n", ori_graph, graph[i * n + j]);
            }
        }
    } else { // 横着的
        int i = thread_x + block_x * b;
        int j = cent_y;
        if(i < n && j < n) {
            int ori_graph = graph[i * n + j];
            graph[i * n + j] = block[thread_x][thread_y];
            if(ori_graph != graph[i * n + j]) {
                printf("ori block not equalxxxx: %d, %d\n", ori_graph, graph[i * n + j]);
            }
        }
    }    
}

__global__ void kernel_stage1(int p, int n, int* graph) {
    int centra_left = p * b;
    int centra_right = min((p + 1) * b, n);

    __shared__ int central_block[b][b]; // 中心块
    
    int thread_x = threadIdx.x; // 块内偏移量
    int thread_y = threadIdx.y;

    int cent_x = thread_x + centra_left; // thread_x:[0, 32)
    int cent_y = thread_y + centra_left; // thread_y: [0, 32)
    if(cent_x < n && cent_y < n) { // 缓存中心块
        // printf("before cent: %d\n", central_block[thread_x][thread_y]);
        central_block[thread_x][thread_y] = graph[cent_x * n + cent_y];
        // printf("after cent: %d\n", central_block[thread_x][thread_y]);
    }
    // printf("first  cent block, centx: %d, centy: %d: %d\n", central_block[thread_x][thread_y], cent_x, cent_y);
    int range = centra_right - centra_left;
    // printGraph(graph, n);
    // printf("range: %d\n", range);
    __syncthreads(); // 同步
    for(int k = 0; k < range; k++) {
        // int ori_cent_block = central_block[thread_x][thread_y];
        // printf("before update, cent block: %d, centk1:%d, centk2:%d\n", central_block[thread_x][thread_y], central_block[thread_x][k], central_block[k][thread_y]);
        central_block[thread_x][thread_y] = min(central_block[thread_x][thread_y], 
            central_block[thread_x][k] + central_block[k][thread_y]);
        // printf("k: %d, block_x: %d, block_y: %d, threadx: %d thready: %d block: %d graph: %d\n", k, cent_x, cent_y,
        // thread_x, thread_y, 
        // central_block[thread_x][thread_y], graph[cent_x * n + cent_y]);
        // if(central_block[thread_x][thread_y] != ori_cent_block) {
        //     printf("ori block: %d, cur block\n", ori_cent_block, central_block[thread_x][thread_y]);
        // }
        __syncthreads(); // 同步
    }
    // printf("\n\n");
    if(cent_x < n && cent_y < n) {
        graph[cent_x * n + cent_y] = central_block[thread_x][thread_y];
        // printf("set back graph: %d, (centx, centy): (%d, %d)\n", graph[cent_x * n + cent_y], cent_x, cent_y);
    }
}

__global__ void kernel_stage3(int p, int n, int* graph) {
    // 拷贝十字块，只需要当前块对应的两个即可
    __shared__ int v_cross_block[b][b]; // 位于十字块竖直方向的
    __shared__ int h_cross_block[b][b]; // 位于十字块水平方向的

    int block_x = blockIdx.x; // 对标centra_left
    int block_y = blockIdx.y;

    if(block_x >= p) block_x += 1; // 跳过中心块
    if(block_y >= p) block_y += 1; // 跳过中心块

    int centra_left = p * b;
    int centra_right = min((p + 1) * b, n);

    int thread_x = threadIdx.x;
    int thread_y = threadIdx.y;

    int v_cross_x = thread_x + centra_left;
    int v_cross_y = thread_y + block_y * b;

    if(v_cross_x < n && v_cross_y < n) {
        v_cross_block[thread_x][thread_y] = graph[v_cross_x * n + v_cross_y];
    }

    int h_cross_x = thread_x + block_x * b;
    int h_cross_y = thread_y + centra_left;

    if(h_cross_x < n && h_cross_y < n) {
        h_cross_block[thread_x][thread_y] = graph[h_cross_x * n + h_cross_y];
    }

    // 拷贝待计算块
    __shared__ int block[b][b];
    
    int i = thread_x + block_x * b;
    int j = thread_y + block_y * b;

    if(i < n && j < n) {
        block[thread_x][thread_y] = graph[i * b + j]; 
    }

    __syncthreads();

    // 更新
    int thread_k = threadIdx.z;
    block[thread_x][thread_y] = min(block[thread_x][thread_y],
         h_cross_block[thread_x][thread_k] + v_cross_block[thread_k][thread_y]);
    
    // 写回
    if(i < n && j < n) {
        graph[i * b + j] = block[thread_x][thread_y]; 
    }

}

__global__ void kernel_stage3_seq(int p, int n, int* graph) { // 顺序遍历k
    // 拷贝十字块，只需要当前块对应的两个即可
    __shared__ int v_cross_block[b][b]; // 位于十字块竖直方向的
    __shared__ int h_cross_block[b][b]; // 位于十字块水平方向的

    int block_x = blockIdx.x; // 对标centra_left
    int block_y = blockIdx.y;

    if(block_x >= p) block_x += 1; // 跳过中心块
    if(block_y >= p) block_y += 1; // 跳过中心块

    int centra_left = p * b;
    int centra_right = min((p + 1) * b, n);

    int thread_x = threadIdx.x;
    int thread_y = threadIdx.y;

    int v_cross_x = thread_x + centra_left;
    int v_cross_y = thread_y + block_y * b;
    if(v_cross_x < n && v_cross_y < n) {
        v_cross_block[thread_x][thread_y] = graph[v_cross_x * n + v_cross_y];
        // printf("p: %d  ", p);
        printf("p %d, threadx, thready:(%d, %d), v_cross_x, v_cross_y: (%d, %d)\n", p, thread_x, thread_y, v_cross_x, v_cross_y);
    }

    int h_cross_x = thread_x + block_x * b;
    int h_cross_y = thread_y + centra_left;
    if(h_cross_x < n && h_cross_y < n) {
        h_cross_block[thread_x][thread_y] = graph[h_cross_x * n + h_cross_y];
        printf("p: %d, threadx, thready:(%d, %d), h_cross_x, h_cross_y: (%d, %d)\n", p, thread_x, thread_y, h_cross_x, h_cross_y);
    }
    __shared__ int block[b][b];
    
    int i = thread_x + block_x * b;
    int j = thread_y + block_y * b;

    if(i < n && j < n) {
        block[thread_x][thread_y] = graph[i * n + j]; 
    }
    // printf("block: %d\n", block[thread_x][thread_y]);

    __syncthreads();
    int range = centra_right - centra_left;
    for(int k = 0; k < range; k++) {
        int ori_blcok = block[thread_x][thread_y];
        block[thread_x][thread_y] = min(block[thread_x][thread_y],
            h_cross_block[thread_x][k] + v_cross_block[k][thread_y]);
        if(ori_blcok != block[thread_x][thread_y]) {
            printf("diff (i, j): %d,, ori_block: %d, new_block: %d, h_cross_block: %d, v_cross_block: %d,  p: %d, k: %d, threadx, thready:(%d, %d), h_cross_x, h_cross_y: (%d, %d)\n", 
            i, j, ori_blcok, block[thread_x][thread_y], h_cross_block[thread_x][k], v_cross_block[k][thread_y], p, k, thread_x, thread_y, h_cross_x, h_cross_y);
        }
    }
    if(i < n && j < n) {
        graph[i * n + j] = block[thread_x][thread_y]; 
    }

}

void printGraph3(int* _graph, int n) {
    for(int j = 0; j < n; j++) {
        for(int i = 0; i < n; i++) {
            std::cout << _graph[i * n + j] << "          ";
        }
        std::cout << std::endl;
    }
}

void apsp(int n, int* graph) {
    std::cout << "Ref start" << std::endl;
    // int* graph2 = (int*)malloc(32 * 32 * sizeof(int));
    // CHK_CUDA_ERR(hipMemcpy(graph2, (void*)graph, sizeof(int) * n * n, hipMemcpyDefault));
    // // // printf("ret: %d\n", ret == CHK_CUDA_ERR);
    
    // printGraph3(graph2, n);
    printf("hipMalloc function : %s\n",hipGetErrorString(hipGetLastError()));
    std::cout << "Ref end\n";
    for(int p = 0; p < (n - 1) / b + 1; p++) {
        // stage 1:
        dim3 thr_1(b, b);
        // dim3 blk_1 = 1;
        kernel_stage1<<<1, thr_1>>>(p, n, graph);
        // stage 2:
        dim3 thr_2(b, b);
        // dim3 blk((n - 1) / 32 + 1, (n - 1) / 32 + 1);
        dim3 blk_2((n - 1) / b, 2); // 所有待更新的block(去掉中心块)
        kernel_stage2<<<blk_2, thr_2>>>(p, n, graph);

        // // stage 3:
        dim3 thr_3(b, b, b); // TODO: 这样是否可行
        dim3 blk_3((n - 1) / b, (n - 1) / b);
        // kernel_stage3<<<blk_3, thr_3>>>(p, n, graph);
        kernel_stage3_seq<<<blk_3, thr_2>>>(p, n, graph);
    }
    // std::cout << "Ref start" << std::endl;
    // int* graph2 = (int*)malloc(32 * 32 * sizeof(int));
    // CHK_CUDA_ERR(hipMemcpy(graph2, (void*)graph, sizeof(int) * n * n, hipMemcpyDefault));
    // // // printf("ret: %d\n", ret == CHK_CUDA_ERR);
    
    // printGraph3(graph2, n);
    // printf("hipMalloc function : %s\n",hipGetErrorString(hipGetLastError()));
    // std::cout << "Ref end\n";
}

